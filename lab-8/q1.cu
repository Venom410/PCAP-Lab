#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 4 // Matrix size
#define NNZ 5 // Number of non-zero elements

__global__ void csrMatrixVectorMul(const int* row_ptr, const int* col_indices, const float* values, const float* x, float* y) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
   
    if (tid < N) {
        float dot_product = 0.0f;
        int row_start = row_ptr[tid];
        int row_end = row_ptr[tid + 1];
       
        for (int j = row_start; j < row_end; j++) {
            dot_product += values[j] * x[col_indices[j]];
        }
       
        y[tid] = dot_product;
    }
}

int main() {
    // Host CSR representation
    int row_ptr[N + 1] = {0, 2, 3, 4,5};
    int col_indices[NNZ] = {0, 2, 3, 1, 2};
    float values[NNZ] = {2.0, 3.0, 4.0, 1.0, 5.0};

    float x[N] = {1.0, 2.0, 3.0, 4.0};
    float y[N]; // Output vector


    // Device data structures
    int *d_row_ptr, *d_col_indices;
    float *d_values, *d_x, *d_y;

    // Allocate memory on the GPU
    hipMalloc((void**)&d_row_ptr, (N + 1) * sizeof(int));
    hipMalloc((void**)&d_col_indices, NNZ * sizeof(int));
    hipMalloc((void**)&d_values, NNZ * sizeof(float));
    hipMalloc((void**)&d_x, N * sizeof(float));
    hipMalloc((void**)&d_y, N * sizeof(float));

    // Copy data from host to device
    hipMemcpy(d_row_ptr, row_ptr, (N + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_col_indices, col_indices, NNZ * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_values, values, NNZ * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice);

    // Define the grid and block dimensions
    dim3 grid(1, 1, 1);
    dim3 block(N, 1, 1);

    // Call the CSR matrix-vector multiplication kernel
    csrMatrixVectorMul<<<grid, block>>>(d_row_ptr, d_col_indices, d_values, d_x, d_y);

    // Copy the result back from the device to the host
    hipMemcpy(y, d_y, N * sizeof(float), hipMemcpyDeviceToHost);

    // Print the result
    printf("Result vector:\n");
    for (int i = 0; i < N; i++) {
        printf("%.2f ", y[i]);
    }
    printf("\n");

    // Free allocated memory on the GPU
    hipFree(d_row_ptr);
    hipFree(d_col_indices);
    hipFree(d_values);
    hipFree(d_x);
    hipFree(d_y);

    return 0;
}

