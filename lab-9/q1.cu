#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 4 // Size of the square matrices

__global__ void matrixMul(int *A, int *B, int *C, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < n && col < n) {
        int sum = 0;
        for (int k = 0; k < n; k++) {
            sum += A[row * n + k] * B[k * n + col];
        }
        C[row * n + col] = sum;
    }
}

int main() {
    int h_A[N][N] = {
        {1, 2, 3, 4},
        {5, 6, 7, 8},
        {9, 10, 11, 12},
        {13, 14, 15, 16}
    };

    int h_B[N][N] = {
        {1, 5, 9, 13},
        {2, 6, 10, 14},
        {3, 7, 11, 15},
        {4, 8, 12, 16}
    };

    int h_C[N][N];

    int *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, N * N * sizeof(int));
    hipMalloc((void **)&d_B, N * N * sizeof(int));
    hipMalloc((void **)&d_C, N * N * sizeof(int));

    // Copy host matrices to device
    hipMemcpy(d_A, h_A, N * N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, N * N * sizeof(int), hipMemcpyHostToDevice);

    // Define grid and block dimensions (2D)
    dim3 dimGrid(N/2, N/2);
    dim3 dimBlock(2, 2);

    // Launch the matrix multiplication kernel
    matrixMul<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, N);

    // Copy the result back to the host
    hipMemcpy(h_C, d_C, N * N * sizeof(int), hipMemcpyDeviceToHost);

    // Print the result matrix
    printf("Result Matrix:\n");
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            printf("%d\t", h_C[i][j]);
        }
        printf("\n");
    }

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
