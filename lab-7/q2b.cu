#include <iostream>
#include <hip/hip_runtime.h>

const int N = 4;
const int M = 4;

// CUDA kernel for matrix multiplication (Each column of the resultant matrix computed by one thread)
__global__ void matrixMultiplyColumn(int *a, int *b, int *c) {
    int col = blockIdx.x; // Each column computed by one thread
    int row = threadIdx.x;

    int result = 0;
    for (int i = 0; i < N; i++) {
        result += a[row * N + i] * b[i * M + col];
    }

    c[row * M + col] = result;
}

int main() {
    int a[N][N], b[N][M], c[N][M];
    int *dev_a, *dev_b, *dev_c;

    // Initialize matrices a and b
    std::cout << "Enter values for matrix A (" << N << "x" << N << "):" << std::endl;
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            std::cin >> a[i][j];
        }
    }

    std::cout << "Enter values for matrix B (" << N << "x" << M << "):" << std::endl;
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < M; j++) {
            std::cin >> b[i][j];
        }
    }

    // Allocate memory on the GPU
    hipMalloc((void**)&dev_a, N * N * sizeof(int));
    hipMalloc((void**)&dev_b, N * M * sizeof(int));
    hipMalloc((void**)&dev_c, N * M * sizeof(int));

    // Copy matrices a and b from host to device
    hipMemcpy(dev_a, a, N * N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N * M * sizeof(int), hipMemcpyHostToDevice);

    // Define the thread and block dimensions
    dim3 blockDim(N, 1);  // Each thread computes one element in a column
    dim3 gridDim(M, 1);   // Grid of size M x 1 (one thread per column)

    // Launch the CUDA kernel for matrix multiplication column-wise
    matrixMultiplyColumn<<<gridDim, blockDim>>>(dev_a, dev_b, dev_c);

    // Copy the result matrix c from device to host
    hipMemcpy(c, dev_c, N * M * sizeof(int), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    // Print the result matrix
    std::cout << "Resultant matrix C:" << std::endl;
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < M; j++) {
            std::cout << c[i][j] << " ";
        }
        std::cout << std::endl;
    }

    return 0;
}
