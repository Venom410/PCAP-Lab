
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 5
#define THREADS_PER_BLOCK 256

__global__ void addVectors(int *a, int *b, int *c, int n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < n) {
        c[tid] = a[tid] + b[tid];
    }
}

int main() {
    int a[N] = {1, 2, 3, 4, 5};  // Initialize host vector 'a'
    int b[N] = {6, 7, 8, 9, 10}; // Initialize host vector 'b'
    int c[N];  // Host result vector
    int *d_a, *d_b, *d_c;   // Device vectors

    // Allocate memory on the device
    hipMalloc((void **)&d_a, N * sizeof(int));
    hipMalloc((void **)&d_b, N * sizeof(int));
    hipMalloc((void **)&d_c, N * sizeof(int));

    // Copy host vectors to device
    hipMemcpy(d_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N * sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel to add vectors
    addVectors<<<(N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(d_a, d_b, d_c, N);

    // Copy result back to host
    hipMemcpy(c, d_c, N * sizeof(int), hipMemcpyDeviceToHost);

    // Cleanup
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // Print result
    printf("Result: ");
    for (int i = 0; i < N; i++) {
        printf("%d ", c[i]);
    }
    printf("\n");

    return 0;
}
