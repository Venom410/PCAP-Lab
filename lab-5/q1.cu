#include <stdio.h>
#include"hip/hip_runtime.h"

#define N 3  // Length of vectors
#define THREADS_PER_BLOCK N

__global__ void addVectorsWithBlockSize(int *a, int *b, int *c) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < N) {
        c[tid] = a[tid] + b[tid];
    }
}

__global__ void addVectorsWithNThreads(int *a, int *b, int *c) {
    int tid = threadIdx.x;
    if (tid < N) {
        c[tid] = a[tid] + b[tid];
    }
}

int main() {
    int a[N] = {1, 3, 5};
    int b[N] = {2, 4, 6};
    int c[N];  // Host result vector
    int *d_a, *d_b, *d_c;   // Device vectors

    // Allocate memory on the device
    hipMalloc((void **)&d_a, N * sizeof(int));
    hipMalloc((void **)&d_b, N * sizeof(int));
    hipMalloc((void **)&d_c, N * sizeof(int));

    // Copy host vectors to device
    hipMemcpy(d_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N * sizeof(int), hipMemcpyHostToDevice);

    // Part A: Using block size as N
    addVectorsWithBlockSize<<<1, THREADS_PER_BLOCK>>>(d_a, d_b, d_c);

    // Part B: Using N threads
    addVectorsWithNThreads<<<1, N>>>(d_a, d_b, d_c);

    // Copy result back to host
    hipMemcpy(c, d_c, N * sizeof(int), hipMemcpyDeviceToHost);

    // Cleanup
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // Print result
    printf("Result: ");
    for (int i = 0; i < N; i++) {
        printf("%d ", c[i]);
    }
    printf("\n");

    return 0;
}
